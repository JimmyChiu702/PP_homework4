/**********************************************************************
 * DESCRIPTION:
 *   Serial Concurrent Wave Equation - C Version
 *   This program implements the concurrent wave equation
 *********************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define MAXPOINTS 1000000
#define MAXSTEPS 1000000
#define MINPOINTS 20
#define PI 3.14159265

void check_param(void);
void init_line(void);
void update(void);
void printfinal8(void);

int nsteps,                 	/* number of time steps */
    tpoints, 	     		/* total points along string */
    rcode;                  	/* generic return code */
float  values[MAXPOINTS+2], 	/* values at time t */
       oldval[MAXPOINTS+2], 	/* values at time (t-dt) */
       newval[MAXPOINTS+2]; 	/* values at time (t+dt) */


/**********************************************************************
 *	Checks input values from parameters
 *********************************************************************/
void check_param(void)
{
   char tchar[20];

   /* check number of points, number of iterations */
   while ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS)) {
      printf("Enter number of points along vibrating string [%d-%d]: "
           ,MINPOINTS, MAXPOINTS);
      scanf("%s", tchar);
      tpoints = atoi(tchar);
      if ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS))
         printf("Invalid. Please enter value between %d and %d\n", 
                 MINPOINTS, MAXPOINTS);
   }
   while ((nsteps < 1) || (nsteps > MAXSTEPS)) {
      printf("Enter number of time steps [1-%d]: ", MAXSTEPS);
      scanf("%s", tchar);
      nsteps = atoi(tchar);
      if ((nsteps < 1) || (nsteps > MAXSTEPS))
         printf("Invalid. Please enter value between 1 and %d\n", MAXSTEPS);
   }

   printf("Using points = %d, steps = %d\n", tpoints, nsteps);

}

/**********************************************************************
 *     Initialize points on line
 *********************************************************************/
void init_line(void)
{
   int i, j;
   float x, fac, k, tmp;

   /* Calculate initial values based on sine curve */
   fac = 2.0 * PI;
   k = 0.0; 
   tmp = tpoints - 1;
   for (j = 1; j <= tpoints; j++) {
      x = k/tmp;
      values[j] = sin(fac * x);
      k = k + 1.0;
   } 

   /* Initialize old values array */
   for (i = 1; i <= tpoints; i++) 
      oldval[i] = values[i];
}

/**********************************************************************
 *      Calculate new values using wave equation
 *********************************************************************/
__device__ void do_math(int idx, float *values_d, float *oldval_d, float *newval_d)
{
   float dtime, c, dx, tau, sqtau;

   dtime = 0.3;
   c = 1.0;
   dx = 1.0;
   tau = (c * dtime / dx);
   sqtau = tau * tau;
   newval_d[idx] = (2.0 * values_d[idx]) - oldval_d[idx] + (sqtau *  (-2.0)*values_d[idx]);
}

/**********************************************************************
 *     Update all values along line a specified number of times
 *********************************************************************/
__global__ void vecUpdate(int *nsteps_d, int *tpoints_d, float *values_d, float *oldval_d, float *newval_d)
{
   int idx = threadIdx.x + 1;

   // Update values for each time step
   int i;
   for (i=1; i<=*nsteps_d; i++) {
      // Update poitns along line for this time step
      if ((idx==1) || (idx==*tpoints_d))
         newval_d[idx] = 0.0;
      else
         do_math(idx, values_d, oldval_d, newval_d);

      oldval_d[idx] = values_d[idx];
      values_d[idx] = newval_d[idx];
   }
}

void update()
{
   int size = (tpoints+2)*sizeof(float);
   int *nsteps_d, *tpoints_d;
   float *values_d, *oldval_d, *newval_d;

   // Transfer nsteps, tpoints, values and oldval to the device
   hipMalloc(&nsteps_d, sizeof(int));
   hipMemcpy(nsteps_d, &nsteps, sizeof(int), hipMemcpyHostToDevice);
   hipMalloc(&tpoints_d, sizeof(int));
   hipMemcpy(tpoints_d, &tpoints, sizeof(int), hipMemcpyHostToDevice);
   hipMalloc(&values_d, size);
   hipMemcpy(values_d, values, size, hipMemcpyHostToDevice);
   hipMalloc(&oldval_d, size);
   hipMemcpy(oldval_d, oldval, size, hipMemcpyHostToDevice);

   // Allocate newval on the device
   hipMalloc(&newval_d, size);

   // Launch device computation threads
   vecUpdate<<<1, tpoints>>>(nsteps_d, tpoints_d, values_d, oldval_d, newval_d);

   // Transfer values back to the host
   hipMemcpy(values, values_d, size, hipMemcpyDeviceToHost);

   // Free device memory
   hipFree(values_d);
   hipFree(oldval_d);
   hipFree(newval_d);
}

/**********************************************************************
 *     Print final results
 *********************************************************************/
void printfinal()
{
   int i;

   for (i = 1; i <= tpoints; i++) {
      printf("%6.4f ", values[i]);
      if (i%10 == 0)
         printf("\n");
   }
}

/**********************************************************************
 *	Main program
 *********************************************************************/
int main(int argc, char *argv[])
{
	sscanf(argv[1],"%d",&tpoints);
	sscanf(argv[2],"%d",&nsteps);
	check_param();
	printf("Initializing points on the line...\n");
	init_line();
	printf("Updating all points for all time steps...\n");
	update();
	printf("Printing final results...\n");
	printfinal();
	printf("\nDone.\n\n");
	
	return 0;
}
